
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define BLOCK_SIZE 64

__global__ void integrate(double *output, const double step, const int steps) {
  const int global_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int local_id = threadIdx.x;
  __shared__ double cache[BLOCK_SIZE];

  // Compute the single value of the integration
  if (global_id < steps) {
    double x = (global_id + 0.5) * step;
    cache[local_id] = 4.0 / (1.0 + x * x);
  } else {
    cache[local_id] = 0;
  }
  __syncthreads();

  // Calculate the sum of the local work group
  for (int offset = BLOCK_SIZE / 2; offset > 0; offset /= 2) {
    if (local_id < offset) {
      cache[local_id] += cache[local_id + offset];
    }
    __syncthreads();
  }

  // Update the cumulative sum of the local group in the global cache
  if (local_id == 0) {
    output[global_id / BLOCK_SIZE] = cache[0];
  }
}

int main(int argc, char *argv[]) {
  unsigned steps = (argc > 1 ? atoi(argv[1]) : 1000000);
  unsigned trials = (argc > 2 ? atoi(argv[2]) : 1000);

  // Initialize the block and grid dimensions
  size_t grid_size, global_size;
  double *results, *output_memory;

  grid_size = (steps + BLOCK_SIZE - 1) / BLOCK_SIZE;
  global_size = BLOCK_SIZE * grid_size;
  double step = 1.0 / (double)(global_size);

  // Initialize the memory buffers on both host and device
  size_t output_memory_bytes = grid_size * sizeof(double);
  results = (double *)malloc(output_memory_bytes);
  hipMalloc((void **)&output_memory, output_memory_bytes);

  // Do a warm up run
  for (unsigned t = 0; t < 100; t++) {
    integrate<<<grid_size, BLOCK_SIZE>>>(output_memory, step, steps);
    hipDeviceSynchronize();
  }

  // Execute the kernel
  double pi;
  clock_t t = clock();
  for (unsigned t = 0; t < trials; t++) {
    integrate<<<grid_size, BLOCK_SIZE>>>(output_memory, step, steps);
    hipDeviceSynchronize();

    // Copy the value from the device to host
    hipMemcpy(results, output_memory, output_memory_bytes,
               hipMemcpyDeviceToHost);

    // Calculate the pi value
    double sum = 0;
    for (int i = 0; i < grid_size; i++)
      sum += results[i];
    pi = step * sum;
  }

  t = clock() - t;
  printf("N: %u local: %i grid: %lu pi: %.3f time: %e\n", steps, BLOCK_SIZE,
         grid_size, pi, (double)t / (CLOCKS_PER_SEC * trials));

  // Release the resources
  free(results);
  hipFree(output_memory);

  return 0;
}
