
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define SHMEM_SIZE 256 * sizeof(double)
#define SIZE 256

__device__ void warp_reduce(volatile double *sdata, unsigned int tid) {
  if (blockDim.x >= 64)
    sdata[tid] += sdata[tid + 32];
  if (blockDim.x >= 32)
    sdata[tid] += sdata[tid + 16];
  if (blockDim.x >= 16)
    sdata[tid] += sdata[tid + 8];
  if (blockDim.x >= 8)
    sdata[tid] += sdata[tid + 4];
  if (blockDim.x >= 4)
    sdata[tid] += sdata[tid + 2];
  if (blockDim.x >= 2)
    sdata[tid] += sdata[tid + 1];
}

__global__ void sum_reduction(double *v, double *v_r, unsigned int n) {

  // Allocate shared memory
  __shared__ double partial_sum[SHMEM_SIZE];

  unsigned int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    partial_sum[tid] = v[i];
  else
    partial_sum[tid] = 0;
  __syncthreads();

  if (blockDim.x >= 512) {
    if (tid < 256) {
      partial_sum[tid] += partial_sum[tid + 256];
    }
    __syncthreads();
  }
  if (blockDim.x >= 256) {
    if (tid < 128) {
      partial_sum[tid] += partial_sum[tid + 128];
    }
    __syncthreads();
  }
  if (blockDim.x >= 128) {
    if (tid < 64) {
      partial_sum[tid] += partial_sum[tid + 64];
    }
    __syncthreads();
  }
  if (tid < 32)
    warp_reduce(partial_sum, tid);

  if (tid == 0)
    v_r[blockIdx.x] = partial_sum[0];
}

int main(int argc, char *argv[]) {
  // vector size
  int n = (argc > 1 ? atoi(argv[1]) : 1 << 16);
  int trials = (argc > 2 ? atoi(argv[2]) : 1000);
  size_t bytes = n * sizeof(double);
  double sequential_sum = 0;

  // TB Size
  int TB_SIZE = SIZE;

  // Grid Size
  int GRID_SIZE = (n + TB_SIZE - 1) / TB_SIZE;

  // Original vector and result vector
  double *h_v, *h_v_r;
  double *d_v, *d_v_r;

  // Allocate memory
  h_v = (double *)malloc(bytes);
  h_v_r = (double *)malloc(sizeof(double) * GRID_SIZE);
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_v_r, bytes);

  // Initialize vector
  for (int i = 0; i < n; i++) {
    h_v[i] = (double)rand();
    sequential_sum += h_v[i];
  }

  // Copy to device
  hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

  // Do a warm up run
  for (unsigned t = 0; t < 100; t++)
    sum_reduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r, n);

  // Time the vector addition
  clock_t t = clock();

  for (unsigned t = 0; t < trials; t++)
    sum_reduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r, n);

  hipDeviceSynchronize();

  // Copy to host;
  hipMemcpy(h_v_r, d_v_r, sizeof(double) * GRID_SIZE, hipMemcpyDeviceToHost);

  double sum = 0;
  for (int i = 0; i < GRID_SIZE; i++)
    sum += h_v_r[i];

  t = clock() - t;

  assert(fabs(sum - sequential_sum) < 1e-12);

  printf("N: %d blockSize: %d gridSize: %d time %e\n", n, TB_SIZE, GRID_SIZE,
         (double)t / (CLOCKS_PER_SEC * trials));

  // Release device memory
  hipFree(d_v);
  hipFree(d_v_r);

  // Release host memory
  free(h_v);
  free(h_v_r);

  return 0;
}
