
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 16

__global__ void matrix_mul(float *a, float *b, float *c, unsigned width_a,
                           unsigned width_b, unsigned height_a) {

  // Get our global thread ID
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < width_b && row < height_a) {
    double dot = 0;

    for (int i = 0; i < width_a; i++)
      dot += a[width_a * row + i] * b[width_b * i + col];

    c[width_b * row + col] = dot;
  }
}

int main(int argc, char *argv[]) {
  // Size of vectors
  unsigned height_a = (argc > 1 ? atoi(argv[1]) : 500);
  unsigned width_a = (argc > 2 ? atoi(argv[2]) : 110);
  unsigned width_b = (argc > 3 ? atoi(argv[3]) : 220);
  unsigned trials = (argc > 2 ? atoi(argv[2]) : 1000);
  unsigned N = height_a * width_b;

  // Host input,outut vectors
  float *h_a, *h_b, *h_c;

  // Device input,output vectors
  float *d_a, *d_b, *d_c;

  // Allocate host memory
  h_a = (float *)malloc(sizeof(float) * width_a * height_a);
  h_b = (float *)malloc(sizeof(float) * width_a * width_b);
  h_c = (float *)malloc(height_a * width_b * sizeof(float));
  // Allocate memory for each vector on GPU
  hipMalloc(&d_a, sizeof(float) * width_a * height_a);
  hipMalloc(&d_b, sizeof(float) * width_a * width_b);
  hipMalloc(&d_c, sizeof(float) * width_b * height_a);

  // Initialize vectors on host
  for (unsigned i = 0; i < height_a * width_a; i++)
    h_a[i] = rand() % 256;

  for (unsigned i = 0; i < width_a * width_b; i++)
    h_b[i] = rand() % 256;

  // Copy host vectors to device
  hipMemcpy(d_a, h_a, sizeof(float) * width_a * height_a,
             hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(float) * width_a * width_b,
             hipMemcpyHostToDevice);

  // Number of thread blocks in grid
  unsigned grid_rows = (height_a + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned grid_cols = (width_b + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dim_grid(grid_cols, grid_rows);
  dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);

  // Do a warm up run
  for (unsigned t = 0; t < 100; t++)
    matrix_mul<<<dim_grid, dim_block>>>(d_a, d_b, d_c, width_a, width_b,
                                        height_a);

  hipDeviceSynchronize();

  // Time the vector addition
  clock_t t = clock();

  // Execute the kernel
  for (unsigned t = 0; t < trials; t++)
    matrix_mul<<<dim_grid, dim_block>>>(d_a, d_b, d_c, width_a, width_b,
                                        height_a);

  // block CPU execution until all previously issued commands on the device have
  // completed
  hipDeviceSynchronize();

  t = clock() - t;
  // Copy array back to host
  hipMemcpy(h_c, d_c, sizeof(float) * height_a * width_b,
             hipMemcpyDeviceToHost);

  for (unsigned i = 0; i < height_a; i++) {
    for (unsigned j = 0; j < width_b; j++) {
      double dot = 0;
      for (unsigned k = 0; k < width_a; k++)
        dot += h_a[i * width_a + k] * h_b[k * width_b + j];
      assert(fabs(h_c[i * width_b + j] - dot) < 1e-8);
    }
  }

  printf("N: %u blockSize: (%d,%d) gridSize: (%d,%d) time %e\n", N, BLOCK_SIZE,
         BLOCK_SIZE, grid_cols, grid_rows,
         (double)t / (CLOCKS_PER_SEC * trials));

  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Release host memory
  free(h_a), free(h_b), free(h_c);
  return 0;
}
